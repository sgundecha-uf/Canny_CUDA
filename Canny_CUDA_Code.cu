#include "hip/hip_runtime.h"
//Group no : 6
//Contact email : karan.magiya@ufl.edu

//Data precision(input and output) : 8 - bit unsigned array with binary values
//Data size range : from 2x2 matrix to 2^15x2^15

//Description of code : Implementing Image processing Canny kernel using Nvidia NPP library. The sobel kernel size is 3x3 and the final step, Non-maximum supression is implemented via custom kernels
//Half of the image is set to '1' value.
//The kernel is then executed for 1000 times. And the kernel execution time is averaged over the 1000 values.

#include <memory>
#include <math.h>
#include <time.h>
#include <iostream>
#include <stdio.h> 
#include <stdlib.h> 
#include <fstream> 
#include <npp.h>
#include <nppi.h>
#include <npps.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <helper_functions.h> // includes for SDK helper functions
#include <hip/hip_runtime_api.h>      // includes for cuda initialization and error checking

using namespace std;

__global__ void sobel(Npp8u *s_x, Npp8u *s_y, Npp8u *d_o, int size) // Calculating the final sobel output
{
	int Idx = threadIdx.x + blockIdx.x * 1024; //Get the thread Id
	float sobel;
	int pointer = Idx * size;
	for(int i = 0; i < size; i++) 
	{
		sobel = sqrt(s_x[pointer]*s[pointer] + s_y[pointer]*s_y[pointer]); 
		d_o[pointer] = sobel;
	}
}

__global__ void canny(Npp8u *d_o, int size) // Non-maximum supression
{
	int Idx = threadIdx.x + blockIdx.x * 1024; //Get the thread Id
	int lowThresh = 50;	// Lower threshold
	int highThresh = 140; // Higher threshold
	int m = sqrt(Idx + 1);  // Get the number elements in one row	

	int pointer = Idx * size;

	//Check for the non-maximum supression
	for(int i = 0; i < size; i++) 
	{
		if(d_o[Idx] > highThresh) 
			d_o[Idx] = 255;
		elseif(d_o[Idx] < lowThresh)
			d_o[Idx] = 0;
		else
		{
			if(d_o[Idx - 1] > highThresh || d_o[Idx + 1] > highThresh || d_o[Idx + m] > highThresh || d_o[Idx - m] > highThresh)
				d_o[Idx] = 255;
			else	
				d_o[Idx] = 0;
		}
	}
}

int main()
{

	//Host and Device Array pointers
	Npp8u * h_i;
	Npp8u * h_o;
	Npp8u * d_i;
	Npp8u * d_o_s_x;	   
	Npp8u * d_o_s_y;
	Npp8u * d_o;
	int * size;	
	//Timing varibales
	hipEvent_t start,stop;
	long double elapsedTime;

	//CREATING EVENTS
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	long double sum[15];

	int count = 0;
	int n;
	for(int m = 0; m < 32767; m++)
	{
		sum[count] = 0;	
		
		//Host and Device memory allocation
		h_i = (Npp8u*)malloc(m * m * sizeof(Npp8u));
		h_o = (Npp8u*)malloc(m * m * sizeof(Npp8u));
		hipMalloc((Npp8u*) &d_i, m * m * sizeof(Npp8u));
		hipMalloc((Npp8u*) &d_o_s_x, m * m * sizeof(Npp8u));
		hipMalloc((Npp8u*) &d_o_s_y, m * m * sizeof(Npp8u));
		hipMalloc((Npp8u*) &d_o, m * m * sizeof(Npp8u));
		hipMalloc((int*) &size, sizeof(int));
		//Initializing the input image data
		int x;
		for (int i = 0; i < m * m; i++)
		{
			h_i[i] = 100;              //Input Gray scale image 
			if (i % m < (m/2))
				h_i[i] = 200;
		}

		// Copying the input image to the Device
		hipMemcpy(d_i, h_i, sizeof(Npp8u) * m * m, hipMemcpyHostToDevice);

		// Copying the size of the input image to the Device
		hipMemcpy(size, m, sizeof(int), hipMemcpyHostToDevice);

		// Defining the number of Blocks and blocks per thread for the custom kernels
		int NUM_BLOCKS, BLOCK_WIDTH;
		BLOCK_WIDTH = m;		
		NUM_BLOCK = m / 1024;
`		if(m % 1024 != 0) // One block can support only 1024 threads. Hence increase number of blocks if needed
			NUM_BLOCK += 1;
		
		NppStatus canny_status;
		
		NppiSize oSrcSize = {m, m}; // ROI of source
		NppiPoint oSrcOffset = {0, 0}; // Offset for the sobel kernel
		
		NppiSize oSizeROI = {m, m};
		
		hipEventRecord(start,0);

		for (int i = 0; i < 1000; i++)
		{
			//hipEventRecord(start,0);
			canny_status = nppiFilterSobelHorizBorder_8u_C1R (d_i, m, oSrcSize, oSrcOffset, d_o_s_x, m, oSizeROI, NPP_BORDER_REPLICATE);  // SObel x
			canny_status = nppiFilterSobelVertBorder_8u_C1R (d_i, m, oSrcSize, oSrcOffset, d_o_s_y, m, oSizeROI, NPP_BORDER_REPLICATE); // SObel y
			sobel<<<NUM_BOCKS,BLOCK_WIDTH>>>(d_o_s_x,d_o_s_y,d_o,size); // Sobel final
			canny<<<NUM_BLOCKS,BLOCK_WIDTH>>>(d_o,size); // Canny final
			
		}
			hipEventRecord(stop,0);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&elapsedTime,start,stop);
			sum[count] = (elapsedTime/1000);		// Final average computed and saved to an array location
		
		//Copy back the output to the host		
		hipMemcpy(h_o, d_o, sizeof(Nppu8) * n * n, hipMemcpyDeviceToHost);

		count++;
		
		//Free the allocated memory
		hipFree(d_o);
		hipFree(d_o_s_x);
		hipFree(d_o_s_y);
		hipFree(d_i);
		hipFree(size);
		Free(h_i);
		Free(h_o);
	}

	ofstream file;
	file.open("resize_up_result_npp.txt");
	for (int i = 0; i < 15; i++)
	{
		file << "Average time taken for 2^" << i << " = " << sum[i] << endl;
	}
	file.close();
	return 0;
}
	
